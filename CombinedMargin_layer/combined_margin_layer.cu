#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cmath>

#include "caffe/layers/combined_margin_layer.hpp"

namespace caffe {

  template <typename Dtype> 
  __global__ void CombinedMarginLayerForward(const int n, const int dim, const Dtype* label_data,
                        Dtype* top_data, Dtype m1, Dtype m2, Dtype m3) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label_data[index]);
      Dtype cos_theta = top_data[index * dim + gt];
      cos_theta = min(cos_theta, Dtype(1.0) - Dtype(1e-4));
      cos_theta = max(cos_theta, Dtype(-1.0) + Dtype(1e-4));
      float theta = acos(cos_theta);
      Dtype m1_mul_theta_plus_m2 = m1 * theta + m2;
      if (m1_mul_theta_plus_m2 > M_PI - Dtype(1e-4)) m1_mul_theta_plus_m2 = M_PI - Dtype(1e-4);
      if (m1_mul_theta_plus_m2 < 0) m1_mul_theta_plus_m2 = 0;
      top_data[index * dim + gt] = cos(m1_mul_theta_plus_m2) -m3;
    }
  }

  template <typename Dtype> 
  __global__ void CombinedMarginLayerBackward(const int n, const int dim, const Dtype* label_data,
                        const Dtype* bottom_data, Dtype* bottom_diff, Dtype m1, Dtype m2) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label_data[index]);
      if(gt < 0) continue;
      Dtype cos_theta = bottom_data[index * dim + gt];
      cos_theta = min(cos_theta, Dtype(1.0) - Dtype(1e-4));
      cos_theta = max(cos_theta, Dtype(-1.0) + Dtype(1e-4));
      float theta = acos(cos_theta);
      Dtype m1_mul_theta_plus_m2 = m1 * theta + m2;
      if (m1_mul_theta_plus_m2 > M_PI - Dtype(1e-4)) m1_mul_theta_plus_m2 = M_PI - Dtype(1e-4);
      if (m1_mul_theta_plus_m2 < 0) m1_mul_theta_plus_m2 = 0;
      Dtype diff_gt = m1 * powf(1 - powf(cos_theta, 2), -0.5) * sin(m1_mul_theta_plus_m2);
      diff_gt = diff_gt > 2 ? 2 : diff_gt;
      diff_gt = diff_gt < 0 ? 0 : diff_gt;
      bottom_diff[index * dim + gt] *= diff_gt;
    }
  }

  template <typename Dtype>
  void CombinedMarginLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                          const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    
    int num = bottom[0]->num();
    int count = bottom[0]->count();
    int dim = count / num;

    caffe_copy(count, bottom_data, top_data);
    
    // NOLINT_NEXT_LINE(whitespace/operators)
    CombinedMarginLayerForward<Dtype> <<<CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >>> (
      num, dim, label_data, top_data, m1, m2, m3);
    CUDA_POST_KERNEL_CHECK;
  }

  template <typename Dtype>
  void CombinedMarginLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                            const vector<bool>& propagate_down,
                            const vector<Blob<Dtype>*>& bottom) {
    
    if (propagate_down[0])
    {
      const Dtype* top_diff = top[0]->gpu_diff();
      const Dtype* label_data = bottom[1]->gpu_data();
      const Dtype* bottom_data = bottom[0]->gpu_data();
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      int count = bottom[0]->count();

      caffe_copy(count, top_diff, bottom_diff);

      int num = bottom[0]->num();
      int dim = count / num;

      // NOLINT_NEXT_LINE(whitespace/operators)
      CombinedMarginLayerBackward<Dtype> <<<CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >>> (
        num, dim, label_data, bottom_data, bottom_diff, m1, m2);
      CUDA_POST_KERNEL_CHECK;
    }
  }
  INSTANTIATE_LAYER_GPU_FUNCS(CombinedMarginLayer);
} // namespace caffe